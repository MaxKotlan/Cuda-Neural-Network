#include "matrix.h"
#include <hipblas.h>
#include "debug.h"

/*Tricks Cublas into Performing Row Major Order Matrix Multiplication using Matrix Transposes*/
void MatrixMultiply(
    uint32_t m, uint32_t k, uint32_t n,
    float alpha, float beta,
    thrust::device_vector<float>& mat_a, 
    thrust::device_vector<float>& mat_b,
    thrust::device_vector<float>& mat_c
){
    DEBUG("MULTIPLYING " << m << "x" << n << " * " << n << "x" << k << " Parameters: Alpha: " << alpha << " Beta: " << beta << std::endl);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(   
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
        n, m, k, 
        &alpha,
        thrust::raw_pointer_cast(mat_b.data()), n,
        thrust::raw_pointer_cast(mat_a.data()), k,
        &beta,
        thrust::raw_pointer_cast(mat_c.data()), n
    );
    hipblasDestroy(handle);
}