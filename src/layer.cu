#include "layer.h"
#include <hipblas.h>
#include "debug.h"

LayerConnector::LayerConnector(uint32_t inputsize, uint32_t outputsize):
    inputsize(inputsize),
    outputsize(outputsize),
    biases(outputsize), 
    weights(inputsize*outputsize),
    d_input(inputsize),
    d_weights(inputsize*outputsize),
    d_biases(outputsize)
{
    InitalizeWithRandomValues();
    thrust::copy(weights.begin(), weights.end(), d_weights.begin());
    thrust::copy(biases.begin(),  biases.end(),  d_biases.begin());
};

void LayerConnector::InitalizeWithRandomValues(){
    float max_range=10.0f;
    for (float &bias : biases)
        bias = max_range*((float)rand() / (float)RAND_MAX)-max_range/2.0f;

    for (float &weight : weights)
        weight = max_range*((float)rand() / (float)RAND_MAX)-max_range/2.0f;// / (float)RAND_MAX;
}

std::vector<float> LayerConnector::operator()(std::vector<float> &previous){
    //previous * weights + bias
    //testssgem();
    DEBUG("LayerConnector: Input - "); for (auto e : previous) DEBUG(e << ", "); DEBUG(std::endl); 
    auto result = CalculateOutputNeurons(previous);
    DEBUG("LayerConnector: Output - "); for (auto e : result) DEBUG(e << ", "); DEBUG(std::endl); 
    return std::move(result);
}

std::vector<float> LayerConnector::CalculateOutputNeurons(std::vector<float>& input){    
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    thrust::device_vector<float> d_output(outputsize);
    thrust::copy(   input.begin(),    input.end(), d_input.begin());
    thrust::copy(d_biases.begin(), d_biases.end(), d_output.begin());

    int m = d_output.size();
    int k = input.size();
    int n = 1;
    float alpha = 1.0;
    float beta  = 1.0; //1.0 because bias vector added and used as output vector
    hipblasSgemm(   
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
        n, m, k, 
        &alpha,
        thrust::raw_pointer_cast(d_input.data())  , n,
        thrust::raw_pointer_cast(d_weights.data()), k,
        &beta,
        thrust::raw_pointer_cast(d_output.data()) , n
    );

    hipblasDestroy(handle);
    std::vector<float> result(biases.size());
    thrust::copy(d_output.begin(), d_output.end(), result.begin());
    return std::move(result);
}