#include "layer.h"
#include <hipblas.h>
#include "debug.h"
#include "activation.h"

LayerConnector::LayerConnector(uint32_t inputsize, uint32_t outputsize):
    inputsize(inputsize),
    outputsize(outputsize),
    biases(outputsize), 
    weights(inputsize*outputsize),
    d_input(inputsize),
    d_weights(inputsize*outputsize),
    d_biases(outputsize)
{
    InitalizeWithRandomValues();
};

void LayerConnector::InitalizeWithRandomValues(){
    float max_range=10.0f;
    for (float &bias : biases)
        bias = max_range*((float)rand() / (float)RAND_MAX)-max_range/2.0f;

    for (float &weight : weights)
        weight = max_range*((float)rand() / (float)RAND_MAX)-max_range/2.0f;// / (float)RAND_MAX;
    
    thrust::copy(weights.begin(), weights.end(), d_weights.begin());
    thrust::copy(biases.begin(),  biases.end(),  d_biases.begin());
}

thrust::device_vector<float> LayerConnector::operator()(thrust::device_vector<float> &d_input){
    //previous * weights + bias
    //testssgem();
    auto result = CalculateOutputNeurons(d_input);
    return std::move(result);
}

thrust::device_vector<float> LayerConnector::CalculateOutputNeurons(thrust::device_vector<float>& d_input_new){    
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    d_input = std::move(d_input_new);

    thrust::device_vector<float> d_output(outputsize);
    thrust::copy(d_biases.begin(), d_biases.end(), d_output.begin());

    int m = d_output.size();
    int k = d_input.size();
    int n = 1;
    float alpha = 1.0;
    float beta  = 1.0; //1.0 because bias vector added and used as output vector
    hipblasSgemm(   
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
        n, m, k, 
        &alpha,
        thrust::raw_pointer_cast(d_input.data())  , n,
        thrust::raw_pointer_cast(d_weights.data()), k,
        &beta,
        thrust::raw_pointer_cast(d_output.data()) , n
    );
    hipblasDestroy(handle);
    thrust::transform(d_output.begin(), d_output.end(), d_output.begin(), Activation::Sigmoid());
    return std::move(d_output);
}

void LayerConnector::CalculateGradient(thrust::device_vector<float>& outputlayer, thrust::device_vector<float>& d_cost){
    thrust::device_vector<float> d_delta_weight(outputsize*inputsize);
    thrust::device_vector<float> d_activation_delta(outputlayer.size());
    d_activation_delta = outputlayer;
    thrust::transform(d_activation_delta.begin(), d_activation_delta.end(), d_activation_delta.begin(), Activation::SigmoidDerivative());
    thrust::transform(d_activation_delta.begin(), d_activation_delta.end(), d_cost.begin(), d_cost.begin(), thrust::multiplies<float>());

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    int m = d_input.size();
    int k = 1;
    int n = d_cost.size();
    float alpha = 1.0;
    float beta  = 0.0;
    hipblasSgemm(   
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
        n, m, k, 
        &alpha,
        thrust::raw_pointer_cast(d_cost.data()),   n,
        thrust::raw_pointer_cast(d_input.data()),  k,
        &beta,
        thrust::raw_pointer_cast(d_delta_weight.data()), n
    );
    hipblasDestroy(handle);

    thrust::copy(d_delta_weight.begin(), d_delta_weight.end(), weights.begin());
    std::cout << std::endl << "Weights: " << std::endl;
    for (auto e : weights){
        std::cout << e << " ";
    }
    std::cout << std::endl;

}
